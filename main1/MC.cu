#include "hip/hip_runtime.h"
/**************************************************************
Lokman A. Abbas-Turki code reused 
by Ly Yannick and Prugniaud Melchior for GPU Programming course of ENSAE 2020

Those who re-use this code should mention in their code 
the name of the author above.
***************************************************************/
#include "rng.h"

#define nt 15
#define nk 6

__constant__ float Tg[nt];
__constant__ float rg[nt];
__constant__ float Kg[nk];
__constant__ float Cg[16*(nt-1)*(nk-1)];

float *Cgc, *Kgc, *Tgc, *rgc;

// Allocate parameters
void VarMalloc()
{
	Kgc = (float *)calloc(nk, sizeof(float));
	Tgc = (float *)calloc(nt, sizeof(float));
	rgc = (float *)calloc(nt, sizeof(float));
	Cgc = (float *)calloc(16*(nk-1)*(nt-1), sizeof(float));
}

// Free parameters
void FreeVar()
{
	free(Cgc);
	free(Kgc);
	free(Tgc);
	free(rgc);
}

// Time parameters
void parameters()
{
	Kgc[0] = 20.f;
	Kgc[1] = 70.f;
	Kgc[2] = 120.f;
	Kgc[3] = 160.f;
	Kgc[4] = 200.f;
	Kgc[5] = 250.0f;

	float d, w, m, y;
	d = 1.0f / 360.0f;
	w = 7.0f * d;
	m = 30.0f * d;
	y = 12.0f * m;
	
	Tgc[0] = d;
 	Tgc[1] = 2.f*d;
 	Tgc[2] = w;
 	Tgc[3] = 2.f*w;
	Tgc[4] = m;
 	Tgc[5] = 2.f*m;
 	Tgc[6] = 3.f*m;
 	Tgc[7] = 6.f*m;
 	Tgc[8] = y;
 	Tgc[9] = y + 3.f*m;
 	Tgc[10] =y + 6.f*m;
 	Tgc[11] = 2.f*y;
 	Tgc[12] = 2.f*y + 6.f*m;
 	Tgc[13] = 3.f*y;
 	Tgc[14] = 3.f*y + 6.f*m;

	rgc[0] = 0.05f;
	rgc[1] = 0.07f;
	rgc[2] = 0.08f;
	rgc[3] = 0.06f;
	rgc[4] = 0.07f;
	rgc[5] = 0.1f;
	rgc[6] = 0.11f;
	rgc[7] = 0.13f;
	rgc[8] = 0.12f;
	rgc[9] = 0.14f;
	rgc[10] = 0.145f;
	rgc[11] = 0.14f;
	rgc[12] = 0.135f;
	rgc[13] = 0.13f;
	rgc[14] = 0.f*y;

	int k;
	FILE *ParFp;
	string TmpString;
	//Spline Volatility parameters------------------------------
	// - Read values from input file on CPU
	TmpString = "Cg.txt";
	ParFp = fopen(TmpString.c_str(),"r");
	if (ParFp == NULL) {
	  fprintf(stderr,"File '%s' unreachable!\n",TmpString.c_str());
	  exit(EXIT_FAILURE);   
	}
	// - Store values in input data tables on CPU
	for (k = 0; k < 1120; k++) {
		if (fscanf(ParFp,"%f",&Cgc[k]) <= 0) {
		  fprintf(stderr,"Error while reading file '%s'!\n",TmpString.c_str());
		  exit(EXIT_FAILURE);          
		}
	}
	fclose(ParFp);
	hipMemcpyToSymbol(HIP_SYMBOL(Kg), Kgc, nk*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(Tg), Tgc, nt*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rg), rgc, nt*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(Cg), Cgc, 16*(nt-1)*(nk-1)*sizeof(float));
}

// Time index  
__device__ int timeIdx(float t) {
	int i, I;
	for (i=14; i>=0; i--) {
		if(t<Tg[i]){
			I = i;
		}
	}
	return I;
}

// Interest rate time integral
__device__ float rt_int(float t,  float T, int i, int j)
{
	float res;
	int k;
	if(i==j){
		res = (T-t)*rg[i];
	}else{
		res = (T-Tg[j-1])*rg[j] + (Tg[i]-t)*rg[i];
		for(k=i+1; k<j; k++){
			res += (Tg[k]-Tg[k-1])*rg[k];
		}
	}

	return res;
}

// Monomials till third degree
__device__ float mon(float x, int i){return 1.0f*(i==0) + x*(i==1) + x*x*(i==2) + x*x*x*(i==3);}

// Local volatility from bicubic interpolation of implied volatility
__device__ void vol_d(float x, float x0, float t, float *V, int q){

	float u1 = 0.0f;
	float u2 = 0.0f;
	float d1, d2, d_1;
	float y = 0.0f;
	float y_1 = 0.0f, y_2 = 0.0f, y_22 = 0.0f;
	int k = 0;
	
	
	if (x >= Kg[5]){
		k = 4;
		d2 = 1.0f /(Kg[k + 1] - Kg[k]);
		u2 = 1.0f;
	}else{
		if (x <= Kg[0]){
			k = 0;
			d2 = 1.0f/(Kg[k + 1] - Kg[k]);
			u2 = 1.0f;
		}else{
			while (Kg[k+1] < x){
				k++;
			}
			d2 = 1.0f/(Kg[k+1] - Kg[k]);
			u2 = (x - Kg[k])/(Kg[k+1] - Kg[k]);
		}
	}

	d1 = 1.0f/(Tg[q + 1] - Tg[q]);
	u1 = (t - Tg[q])/(Tg[q + 1] - Tg[q]);

	for (int i = 0; i < 4; i++){
		for (int j = 0; j < 4; j++){
			y += Cg[k * 14 * 16 + q * 16 + j + i * 4] * mon(u1, i)*mon(u2, j);
			y_1 += i *Cg[k * 14 * 16 + q * 16 + i * 4 + j] * mon(u1, i-1)*mon(u2, j)*d1;
			y_2 += j*Cg[k * 14 * 16 + q * 16 + i * 4 + j] * mon(u1, i)*mon(u2, j-1)*d2;
			y_22 += j *(j - 1)*Cg[k * 14 * 16 + q * 16 + i * 4 + j] * mon(u1, i)*mon(u2, j-2)*d2*d2;
		}
	}
	d_1 = (logf(x0/x) + rt_int(0.0f, t, 0, q))/(y*sqrtf(t)) + 0.5f*y*sqrtf(t);
	u1 = x*x*(y_22 - d_1*sqrtf(t)*y_2*y_2 + (1.0f/y)*((1.0f/(x*sqrtf(t))) 
		+ d_1*y_2)*((1.0f /(x*sqrtf(t))) + d_1*y_2));
	u2 = 2.0f*y_1 + y /t + 2.0f*x*rg[q]*y_2;
	
	*V = sqrtf(fminf(fmaxf(u2/u1,0.0001f),0.5f));
}

// Set the new RNG seed
__device__ void CMRG_set_d(int *a0, int *a1, int *a2, int *a3, int *a4, 
			         int *a5, int *CMRG_Out){
	CMRG_Out[0] = *a0;
	CMRG_Out[1] = *a1;
	CMRG_Out[2] = *a2;
	CMRG_Out[3] = *a3;
	CMRG_Out[4] = *a4;
	CMRG_Out[5] = *a5;
}

// Get the RNG Seed
__device__ void CMRG_get_d(int *a0, int *a1, int *a2, int *a3, int *a4, 
			         int *a5, int *CMRG_In){
	*a0 = CMRG_In[0];
	*a1 = CMRG_In[1];
	*a2 = CMRG_In[2];
	*a3 = CMRG_In[3];
	*a4 = CMRG_In[4];
	*a5 = CMRG_In[5];
}

// Generate uniformly distributed random variables
__device__ void CMRG_d(int *a0, int *a1, int *a2, int *a3, int *a4, 
			     int *a5, float *g0, float *g1, int nb){

 const int m1 = 2147483647;// Requested for the simulation
 const int m2 = 2145483479;// Requested for the simulation
 int h, p12, p13, p21, p23, k, loc;// Requested local parameters

 for(k=0; k<nb; k++){

	 // First Component 
	 h = *a0/q13; 
	 p13 = a13*(h*q13-*a0)-h*r13;
	 h = *a1/q12; 
	 p12 = a12*(*a1-h*q12)-h*r12;

	 if (p13 < 0) {
	   p13 = p13 + m1;
	 }
	 if (p12 < 0) {
	   p12 = p12 + m1;
	 }
	 *a0 = *a1;
	 *a1 = *a2;
	 if( (p12 - p13) < 0){
	   *a2 = p12 - p13 + m1;  
	 } else {
	   *a2 = p12 - p13;
	 }
  
	 // Second Component 
	 h = *a3/q23; 
	 p23 = a23*(h*q23-*a3)-h*r23;
	 h = *a5/q21; 
	 p21 = a21*(*a5-h*q21)-h*r21;

	 if (p23 < 0){
	   p23 = p23 + m2;
	 }
	 if (p12 < 0){
	   p21 = p21 + m2;
	 }
	 *a3 = *a4;
	 *a4 = *a5;
	 if ( (p21 - p23) < 0) {
	   *a5 = p21 - p23 + m2;  
	 } else {
	   *a5 = p21 - p23;
	 }

	 // Combines the two MRGs
	 if(*a2 < *a5){
		loc = *a2 - *a5 + m1;
	 }else{loc = *a2 - *a5;} 

	 if(k){
		if(loc == 0){
			*g1 = Invmp*m1;
		}else{*g1 = Invmp*loc;}
	 }else{
		*g1 = 0.0f; 
		if(loc == 0){
			*g0 = Invmp*m1;
		}else{*g0 = Invmp*loc;}
	 }
  }
}

// Generates Gaussian distribution from a uniform one (Box-Muller)
__device__ void BoxMuller_d(float *g0, float *g1){
  float loc;
  if (*g1 < 1.45e-6f){
    loc = sqrtf(-2.0f*logf(0.00001f))*cosf(*g0*2.0f*MoPI);
  } else {
    if (*g1 > 0.99999f){
      loc = 0.0f;
    } else {loc = sqrtf(-2.0f*logf(*g1))*cosf(*g0*2.0f*MoPI);}
  }
  *g0 = loc;
}

// Euler for local volatility
__device__ void Euler_d(float *S2, float S1, float r0,
						float sigma, float dt, float e){
  *S2 = S1*(1.0f + r0*dt*dt + sigma*dt*e);
}

// MC for inner trajectories
__global__ void MC_inner_k(int P1, int P2, float St, float _t, int It, float dt, float B, float K, int L, int M, int Ntraj, TabSeedCMRG_t *pt_cmrg, float *option_price){

	int gb_index_x = threadIdx.x + blockIdx.x*blockDim.x;
	int a0, a1, a2, a3, a4, a5, k, i, q, P;
	float g0, g1, Sk, Skp1, t, v;
	extern __shared__ float Z[];

	Sk = St;
	P = It;

	CMRG_get_d(&a0, &a1, &a2, &a3, &a4, &a5, pt_cmrg[0][gb_index_x]);

	for (k=int(_t * M); k<M; k++){
		for (i=1; i<=L; i++){
			t = dt*dt*(i+L*k);
			q = timeIdx(t);
			vol_d(Sk, St, t, &v, q);
			CMRG_d(&a0, &a1, &a2, &a3, &a4, &a5, &g0, &g1, 2);
			BoxMuller_d(&g0, &g1);
			Euler_d(&Skp1, Sk, rg[q], v, dt, g0);
			Sk = Skp1;  
		}
		P += (Sk<B);
	}

	// Reduction phase
	Z[threadIdx.x] = expf(-rt_int(_t, t, 0, q))*fmaxf(0.0f, Sk-K)*((P<=P2)&&(P>=P1))/Ntraj;
	Z[threadIdx.x + blockDim.x] = Ntraj*Z[threadIdx.x]*Z[threadIdx.x];
	__syncthreads();

	i = blockDim.x/2;
	while (i != 0) {
		if (threadIdx.x < i){
		Z[threadIdx.x] += Z[threadIdx.x + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0){
		atomicAdd(option_price, Z[0]);
	}

	CMRG_set_d(&a0, &a1, &a2, &a3, &a4, &a5, pt_cmrg[0][gb_index_x]);
}

// MC for outer trajectories
__global__ void MC_outer_k(int P1, int P2, float x_0, float dt, float B, float K, int L, int M, int Ntraj, TabSeedCMRG_t *pt_cmrg, float *option_price, float *sum, float *option_prices, float *stocks, int *It){

  int gb_index_x = threadIdx.x + blockIdx.x*blockDim.x;
  int a0, a1, a2, a3, a4, a5, k, i, q, P, increment;
  float g0, g1, Sk, Skp1, t, v;

  extern __shared__ float H[];

  Sk = x_0;
  P = 0;

  CMRG_get_d(&a0, &a1, &a2, &a3, &a4, &a5, pt_cmrg[0][gb_index_x]);

  for (k=1; k<=M; k++){
	  for (i=1; i<=L; i++){
		  t = dt*dt*(i+L*k);
		  q = timeIdx(t);
		  vol_d(Sk, x_0, t, &v, q);
		  CMRG_d(&a0, &a1, &a2, &a3, &a4, &a5, &g0, &g1, 2);
		  BoxMuller_d(&g0, &g1);
		  Euler_d(&Skp1, Sk, rg[q], v, dt, g0);
		  Sk = Skp1;  
	  }
	  P += (Sk<B);
	increment = k + M * gb_index_x - 1;
	stocks[increment] = Sk;
	It[increment] = P;
	MC_inner_k<<<32,32,2*32*sizeof(float)>>>(P1, P2, Sk, k*dt*dt, P, dt, B, K, L, M, Ntraj, pt_cmrg, option_prices + increment);
  }
  
  // Reduction phase
  H[threadIdx.x] = expf(-rt_int(0.0f, t, 0, q))*fmaxf(0.0f, Sk-K)*((P<=P2)&&(P>=P1))/Ntraj;
  H[threadIdx.x + blockDim.x] = Ntraj*H[threadIdx.x]*H[threadIdx.x];
  __syncthreads();

  i = blockDim.x/2;
  while (i != 0) {
	if (threadIdx.x < i){
		H[threadIdx.x] += H[threadIdx.x + i];
		H[threadIdx.x + blockDim.x] += H[threadIdx.x + blockDim.x + i];
	}
	__syncthreads();
	i /= 2;
  }

  if (threadIdx.x == 0){
	atomicAdd(option_price, H[0]);
	atomicAdd(sum, H[blockDim.x]);
  }

  CMRG_set_d(&a0, &a1, &a2, &a3, &a4, &a5, pt_cmrg[0][gb_index_x]);
}


int main()
{	
	int ti = 1;
	float T = 1.0f;
	float K = 100.0f;
	float S0 = 100.0f;
	float B = 120.0f;
	int M = 100;
	int P1 = 10;
	int P2 = 49;
	int Nt = 200;
	float dt = sqrtf(T/Nt);
	int leng = Nt/M;
	float option_price_CPU = 0.0f;	
	float sum_CPU = 0.0f;
	float Tim;							// GPU timer instructions
	hipEvent_t start, stop;			// GPU timer instructions
	float *option_price_GPU, *sum_GPU, *option_prices_GPU, *option_prices_CPU, *St_GPU, *St_CPU;
	int *It_GPU, *It_CPU;
	int Ntraj = 32*32;

	// Allocation of memories inside GPU 
	hipMalloc(&option_price_GPU, sizeof(float));
	hipMalloc(&sum_GPU, sizeof(float));
	hipMalloc(&St_GPU, Ntraj * M * sizeof(float));
	hipMalloc(&It_GPU, Ntraj * M * sizeof(int));	
	hipMalloc(&option_prices_GPU, Ntraj * M * sizeof(float));
	VarMalloc();

	// Init values
	hipMemset(option_price_GPU, 0.0f, sizeof(float));
	hipMemset(sum_GPU, 0.0f, sizeof(float));
	hipMemset(option_prices_GPU, 0, Ntraj * M * sizeof(float));

	// Allocation of memories inside CPU
	option_prices_CPU = (float*)malloc(Ntraj * M * sizeof(float));
	It_CPU = (int*)malloc(Ntraj*M*sizeof(int));
	St_CPU = (float*)malloc(Ntraj*M*sizeof(float));

	// Init CMRG
	PostInitDataCMRG();
	parameters();

	// GPU timer instructions initialization
	hipEventCreate(&start);			
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	MC_outer_k<<<32,32,2*32*sizeof(float)>>>(P1, P2, S0, dt, B, K, leng, M, Ntraj, CMRG, option_price_GPU, sum_GPU, option_prices_GPU, St_GPU, It_GPU);

	// Transfer data from device (GPU) to host (CPU)
	hipMemcpy(option_prices_CPU, option_prices_GPU, Ntraj * M * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(St_CPU, St_GPU, Ntraj * M * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(It_CPU, It_GPU, Ntraj * M * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&option_price_CPU, option_price_GPU, sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(&sum_CPU, sum_GPU, sizeof(float), hipMemcpyDeviceToHost);

	// GPU timer instructions stop the record
	hipEventRecord(stop,0);			
	hipEventSynchronize(stop);			
	hipEventElapsedTime(&Tim,start,stop);				
	hipEventDestroy(start);			
	hipEventDestroy(stop);				

	// Result of outer trajectories
	printf("The price is equal to %f\n", option_price_CPU);
	printf("error associated to a confidence interval of 95%% = %f\n", 
		1.96*sqrt((double)(1.0f/(Ntraj-1))*(Ntraj*sum_CPU-(option_price_CPU*option_price_CPU)))/sqrt((double)Ntraj));
	printf("Execution time %f ms\n", Tim);
	
	// Writing results is CSV file of inner trajectories
	printf("===================================\nWriting into a csv file inside the current folder\n===================================");
	FILE *outfile = fopen("data_generated.csv","w");
	fprintf(outfile,"Temps, It, Stocks, Prix\n");
	for (int k = 0; k < Ntraj*M; ++k){
		fprintf(outfile,"%i,%i,%f,%f\n", ti, It_CPU[k], St_CPU[k], option_prices_CPU[k]);
		if (ti >= 100)
		{
			ti=0;
		}
		ti++;
	}
	fclose(outfile);

	// Free memory
	FreeCMRG();
	FreeVar();
	hipFree(option_prices_GPU) ; 
	hipFree(St_GPU);
	hipFree(It_GPU);
	hipFree(option_price_GPU);
	hipFree(sum_GPU);
	free(St_CPU);
	free(It_CPU);
	free(option_prices_CPU);
	return 0;
}

// nvcc -arch=sm_35 -rdc=true MC.cu rng.cu -o MC


